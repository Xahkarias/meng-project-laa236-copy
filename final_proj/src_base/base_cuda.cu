#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "../defines.hpp"

// This kernel fills the dp matrix based on 3 of its neighbors and penalties
__global__ void smith_waterman_kernel_optimized(
    const char *__restrict__ seq1,
    const char *__restrict__ seq2,
    int *__restrict__ score,
    int size1,
    int size2,
    int diag)
{
    // shared memory used by threads in a block
    extern __shared__ int shared_score[];
    // current thread parameters
    int thread_id = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned long long i = diag - global_idx;
    unsigned long long j = global_idx + 1;

    // checking bounds
    if (i < 1 || i > size1 || j < 1 || j > size2) return;

    // Compute indices
    unsigned long long index = i * (size2 + 1) + j;
    unsigned long long index_diag = (i - 1) * (size2 + 1) + (j - 1);
    unsigned long long index_up = (i - 1) * (size2 + 1) + j;
    unsigned long long index_left = i * (size2 + 1) + (j - 1);

    // checking miss MATCH_SCORE or MATCH_SCORE
    char a = seq1[i - 1];
    char b = seq2[j - 1];
    int matchScore = (a == b) ? MATCH_SCORE : MISMATCH_SCORE;

    // Use shared memory to store neighboring values
    shared_score[thread_id] = score[index_diag];

    __syncthreads();

    // computing all the possibilites, removed conditional statements for speed up
    int score_diag = shared_score[thread_id] + matchScore;
    int score_up = score[index_up] + GAP_SCORE;
    int score_left = score[index_left] + GAP_SCORE;

    //updating the matrix
    int cellScore = max(0, max(score_diag, max(score_up, score_left)));
    score[index] = cellScore;
}

// Find the maximum score and its position in the score matrix
__global__ void find_max_score_kernel(
    int *__restrict__ score,
    int *__restrict__ max_i, 
    int *__restrict__ max_j,
    int *__restrict__ max_score,
    int size1,
    int size2)
{
    // Shared memory for reduction
    extern __shared__ int shared_data[];
    int *shared_scores = shared_data;
    int *shared_i = &shared_data[blockDim.x];
    int *shared_j = &shared_data[2 * blockDim.x];
    
    int tid = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Thread-local max
    int local_max_score = 0;
    int local_max_i = 0;
    int local_max_j = 0;
    
    // Process multiple elements per thread
    for (int idx = global_idx; idx < (size1 + 1) * (size2 + 1); idx += stride) {
        int i = idx / (size2 + 1);
        int j = idx % (size2 + 1);
        int current_score = score[idx];
        
        if (current_score > local_max_score) {
            local_max_score = current_score;
            local_max_i = i;
            local_max_j = j;
        }
    }
    
    // Load thread-local max into shared memory
    shared_scores[tid] = local_max_score;
    shared_i[tid] = local_max_i;
    shared_j[tid] = local_max_j;
    
    __syncthreads();
    
    // Reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared_scores[tid] < shared_scores[tid + s]) {
                shared_scores[tid] = shared_scores[tid + s];
                shared_i[tid] = shared_i[tid + s];
                shared_j[tid] = shared_j[tid + s];
            }
        }
        __syncthreads();
    }
    
    // Write block's result to global memory
    if (tid == 0) {
        int block_max_score = shared_scores[0];
        atomicMax(max_score, block_max_score);
        
        // Use atomicCAS to update max_i and max_j only if this block has the max score
        int old_max = atomicCAS(max_score, block_max_score, block_max_score);
        if (old_max == block_max_score) {
            atomicExch(max_i, shared_i[0]);
            atomicExch(max_j, shared_j[0]);
        }
    }
}

// Traceback kernel to generate alignment paths
__global__ void traceback_kernel(
    const char *__restrict__ seq1,
    const char *__restrict__ seq2,
    int *__restrict__ score,
    int max_i,
    int max_j,
    int size1,
    int size2,
    char *__restrict__ aligned_seq1,
    char *__restrict__ aligned_seq2,
    int *__restrict__ align_length)
{
    // Only one thread does the traceback
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int i = max_i;
        int j = max_j;
        int idx = 0;
        
        // Temporary arrays to store alignment (will be reversed later)
        char temp_seq1[10000]; // Assuming max length, adjust as needed
        char temp_seq2[10000];
        
        while (i > 0 && j > 0 && score[i * (size2 + 1) + j] > 0) {
            int current_idx = i * (size2 + 1) + j;
            int diag_idx = (i - 1) * (size2 + 1) + (j - 1);
            int up_idx = (i - 1) * (size2 + 1) + j;
            //int left_idx = i * (size2 + 1) + (j - 1);
            
            char a = seq1[i - 1];
            char b = seq2[j - 1];
            int match_score = (a == b) ? MATCH_SCORE : MISMATCH_SCORE;
            
            if (score[current_idx] == score[diag_idx] + match_score) {
                // Diagonal
                temp_seq1[idx] = seq1[i - 1];
                temp_seq2[idx] = seq2[j - 1];
                i--; j--;
            } else if (score[current_idx] == score[up_idx] + GAP_SCORE) {
                // Up
                temp_seq1[idx] = seq1[i - 1];
                temp_seq2[idx] = '-';
                i--;
            } else {
                // Left
                temp_seq1[idx] = '-';
                temp_seq2[idx] = seq2[j - 1];
                j--;
            }
            idx++;
        }
        
        // Reverse the alignment
        for (int k = 0; k < idx; k++) {
            aligned_seq1[k] = temp_seq1[idx - k - 1];
            aligned_seq2[k] = temp_seq2[idx - k - 1];
        }
        
        // Set null terminators
        aligned_seq1[idx] = '\0';
        aligned_seq2[idx] = '\0';
        
        // Set alignment length
        *align_length = idx;
    }
}

std::pair<std::string, std::string> smithWaterman(
    const char *seq1,
    size_t size1,
    const char *seq2,
    size_t size2)
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("Running on GPU %d: %s\n", device, prop.name);
    
    char *cuda_seq1, *cuda_seq2;
    int *cuda_score;
    int *cuda_max_i, *cuda_max_j, *cuda_max_score;
    char *cuda_aligned_seq1, *cuda_aligned_seq2;
    int *cuda_align_length;

    // Allocate memory on the device
    hipMalloc((void **)&cuda_seq1, size1 * sizeof(char));
    hipMalloc((void **)&cuda_seq2, size2 * sizeof(char));
    hipMalloc((void **)&cuda_score, (size1 + 1) * (size2 + 1) * sizeof(int));
    hipMalloc((void **)&cuda_max_i, sizeof(int));
    hipMalloc((void **)&cuda_max_j, sizeof(int));
    hipMalloc((void **)&cuda_max_score, sizeof(int));
    
    // Allocate memory for alignment results (assuming max length)
    size_t max_align_len = size1 + size2;  // Worst case
    hipMalloc((void **)&cuda_aligned_seq1, (max_align_len + 1) * sizeof(char));
    hipMalloc((void **)&cuda_aligned_seq2, (max_align_len + 1) * sizeof(char));
    hipMalloc((void **)&cuda_align_length, sizeof(int));

    // Initialize max values
    hipMemset(cuda_max_i, 0, sizeof(int));
    hipMemset(cuda_max_j, 0, sizeof(int));
    hipMemset(cuda_max_score, 0, sizeof(int));
    
    // Copy sequences to device
    hipMemcpy(cuda_seq1, seq1, size1 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2, seq2, size2 * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(cuda_score, 0, (size1 + 1) * (size2 + 1) * sizeof(int));

    // Fill score matrix in wave-front (anti-diagonal order)
    int total_diagonals = size1 + size2 - 1;
    int threads_per_block = 1024; 
    int shared_mem_size = threads_per_block * sizeof(int);

    for (int diag = 1; diag <= total_diagonals; ++diag) {
        int elements_in_diag = min(diag, min(static_cast<int>(size1), static_cast<int>(size2)));
        int blocks = (elements_in_diag + threads_per_block - 1) / threads_per_block;

        smith_waterman_kernel_optimized<<<blocks, threads_per_block, shared_mem_size>>>(
            cuda_seq1, cuda_seq2, cuda_score,
            size1, size2, diag);
        
        hipDeviceSynchronize();
    }

    // Find maximum score and position using GPU
    int num_blocks = min(32, (int)((size1 * size2 + threads_per_block - 1) / threads_per_block));
    int find_max_shared_mem = threads_per_block * 3 * sizeof(int); // For scores, i indices, and j indices
    
    find_max_score_kernel<<<num_blocks, threads_per_block, find_max_shared_mem>>>(
        cuda_score, cuda_max_i, cuda_max_j, cuda_max_score, 
        size1, size2);
    
    hipDeviceSynchronize();
    
    // Get max values from device
    int max_i, max_j, max_score;
    hipMemcpy(&max_i, cuda_max_i, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max_j, cuda_max_j, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max_score, cuda_max_score, sizeof(int), hipMemcpyDeviceToHost);
    
    // Perform traceback on GPU
    traceback_kernel<<<1, 1>>>(
        cuda_seq1, cuda_seq2, cuda_score,
        max_i, max_j, size1, size2,
        cuda_aligned_seq1, cuda_aligned_seq2, cuda_align_length);
    
    hipDeviceSynchronize();
    
    // Copy alignment results back to host
    int align_length;
    hipMemcpy(&align_length, cuda_align_length, sizeof(int), hipMemcpyDeviceToHost);
    
    std::vector<char> h_aligned_seq1(align_length + 1);
    std::vector<char> h_aligned_seq2(align_length + 1);
    
    hipMemcpy(h_aligned_seq1.data(), cuda_aligned_seq1, (align_length + 1) * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(h_aligned_seq2.data(), cuda_aligned_seq2, (align_length + 1) * sizeof(char), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(cuda_seq1);
    hipFree(cuda_seq2);
    hipFree(cuda_score);
    hipFree(cuda_max_i);
    hipFree(cuda_max_j);
    hipFree(cuda_max_score);
    hipFree(cuda_aligned_seq1);
    hipFree(cuda_aligned_seq2);
    hipFree(cuda_align_length);

    // Convert to strings
    std::string alignedSeq1(h_aligned_seq1.data(), align_length);
    std::string alignedSeq2(h_aligned_seq2.data(), align_length);
    
    return {alignedSeq1, alignedSeq2};
}
